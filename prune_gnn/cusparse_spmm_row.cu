#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdexcept>
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <chrono>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>

#include <torch/extension.h>

using namespace std;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        throw std::runtime_error("error");                                     \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        throw std::runtime_error("error");                                     \
    }                                                                          \
}

torch::Tensor cusparse_spmm_row(torch::Tensor input,
                          torch::Tensor row_pointer,
                          torch::Tensor column_index,
                          torch::Tensor degrees)
{
    const int NODE_NUM = input.size(0);
    const int dim = input.size(1);
    const int nnz = column_index.size(0);
    const int R_PTR_NUM = NODE_NUM + 1;

    input = input.reshape({NODE_NUM * dim, 1});

    // Create output tensor
    torch::Tensor output = torch::zeros(NODE_NUM * dim, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    
    int   A_num_rows = NODE_NUM;
    int   A_num_cols = NODE_NUM;
    int   A_nnz = nnz;
    int   B_num_rows = NODE_NUM;
    int   B_num_cols = dim;

    int   A_num_off       = R_PTR_NUM;
    int   ldb             = NODE_NUM;
    int   ldc             = NODE_NUM;
    int   B_size          = ldb * dim;
    int   C_size          = ldc * dim;


    float alpha           = 1.0f;
    float beta            = 0.0f;

    // Get GPU data pointers
    auto d_row_pointer = row_pointer.data_ptr();
    auto d_column_index = column_index.data_ptr();
    auto d_degrees = degrees.data_ptr();

    auto d_output = output.data_ptr();
    auto d_input = input.data_ptr();

   // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      d_row_pointer, d_column_index, d_degrees,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, dim, d_input,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, dim, d_output,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )



   // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )


  
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

  
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------


    return output.reshape({NODE_NUM, dim});

}
