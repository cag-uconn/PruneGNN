#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>
#include <fstream>
#include <string>
#include <sstream>

// #define FP16MM

#include <torch/extension.h>


const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline
hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
void CPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	int a=1;

    for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		A[i] = (float)rand()/(float)(RAND_MAX/a);
	}
}

torch::Tensor cublas_gemm(torch::Tensor A,
                          torch::Tensor B)
{
    // using namespace std;
    
    int m = A.size(0);
    int k = A.size(1);
    int n = B.size(1);

    A = A.reshape({m * k, 1});
    B = B.reshape({k * n, 1});


    // Create output tensor
    torch::Tensor output = torch::zeros(m * n, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    // Get GPU data pointers

    auto d_A = (float*) A.data_ptr();
    auto d_B = (float*) B.data_ptr();
    auto d_output = (float*) output.data_ptr();


    hipblasStatus_t stat;
    hipblasHandle_t handle;

    checkCublas(hipblasCreate(&handle));


    int lda, ldb, ldc;
    const float alf = 1.0f;
    const float bet = 0.0f;
    const float *alpha = &alf;
    const float *beta = &bet;


	  lda = m;
	  ldb = k;
	  ldc = m;

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_output, ldc); 

    // hipEventRecord(stop,0);
    // hipEventSynchronize(stop);
    if(stat != HIPBLAS_STATUS_SUCCESS){
        // cerr << "hipblasSgemmBatched failed" << std::endl;
        exit(1);
    }
  
    assert(!hipGetLastError());
  
    // //Free GPU memory
    // hipFree(d_A);
    // hipFree(d_B);
    // hipFree(d_output);

    return output.reshape({m, n});

}